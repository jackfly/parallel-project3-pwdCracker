#include "hip/hip_runtime.h"

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <iostream>
#include <algorithm>
#include <stdint.h>
#include "timing.h"
#include <inttypes.h>
#include <time.h>

// includes CUDA
#include <hip/hip_runtime.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper functions for SDK examples
//const int N = 16;
//const int blocksize = 16;
/*
__global__
void hello(char *a, int *b)
{
  a[threadIdx.x] += b[threadIdx.x];
}*/

# define my_sizeof(type) ((char *)(&type+1)-(char*)(&type))
//static int crackFlag = 0;

static uint8_t alphanumerical[36] = {
    97,98,99,100,101,102,103,104,105,106,107,108,109,110,111,112,113,114,115,116,117,118,119,120,121,122,49,50,51,52,53,54,55,56,57,58
};
//"abcdefghijklmnopqrstuvwxyz"
//"0123456789"
static uint8_t password[5] = {
    98,118,51,55,113
  //58
};
//bv37q

//http://www.cse.yorku.ca/~oz/hash.html
//sdbm hash function
uint64_t inline hash(uint8_t str[], int len)
{
    uint64_t hash = 0;
    int c;
    int i = 0;

    for(i = 0; i< len; i++){
        c = str[i];
        hash = c + (hash << 6) + (hash << 16) - hash;
    }
    return hash;
}
__device__
uint64_t inline cudahash(uint8_t str[], int len)
{
    uint64_t hash = 0;
    int c;
    int i = 0;

    for(i = 0; i< len; i++){
        c = str[i];
        hash = c + (hash << 6) + (hash << 16) - hash;
    }
    return hash;
}
__device__
void permutation(uint8_t prefix[], int length, int index, uint64_t *cudahashPassword, int constLength, int output[], uint8_t *cudaalphanumerical){
  //  if (crackFlag == 1) return; // have been cracked password
  if(length == 0){
    //if(prefix[0]==98 && prefix[1] == 104){
    //printf("%d %d %d %d %d\n",prefix[0],prefix[1],prefix[2],prefix[3],prefix[4]);
    //}
  if(cudahash(prefix,constLength)==cudahashPassword[0]){
    printf("======Successfully Cracked!!!======\n");
/*    const int lastone = 5;
    output[lastone] = constLength;
    for (int n = 0; n<constLength; n++){
      output[n] = prefix[n];
    }*/
    printf("The cracked password in ASCII is: ");
        for(int j = 0; j < constLength; j++){
            printf("%d ", prefix[j]);
        }
        printf("\n");
//        crackFlag = 1;
   }
   return;
  }else{
    //for (int i = 0; i < sizeof(cudaalphanumerical)/sizeof(int8_t); i++) {
    for (int i = 0; i < 36; i++) {
      prefix[index] = cudaalphanumerical[i];
      permutation(prefix, length-1,index+1, cudahashPassword, constLength, output, cudaalphanumerical);
    }

  }
 }

__global__
void cudaPermutation(int length, int index, uint64_t hashPassword[], int constLength, int output[], uint8_t *cudaalphanumerical){
  uint8_t prefix[5] = {0};
  prefix[0] = cudaalphanumerical[threadIdx.x];
  //for(int i =0; i<36;i++)
  //printf("%d ", cudaalphanumerical[i]);
  permutation(prefix,length-1,1,hashPassword,length,output,cudaalphanumerical);
}

int main()
{
  int maxLen = 5;
  int *output = new int[maxLen+1];
  uint64_t hashPassword = ::hash(password,my_sizeof(password)/my_sizeof(password[0]));
  //printf("CPU hash: %" PRIu64 "\n", hashPassword);
  const size_t poolsize = 36*sizeof(uint8_t);
  const size_t hashPwdsize = sizeof(uint64_t);
  uint8_t *cudaalphanumerical;
  uint64_t *cudahashPassword;
  hipMalloc( &cudaalphanumerical, poolsize );
  hipMalloc( &cudahashPassword, hashPwdsize );
  hipMemcpy( cudaalphanumerical, alphanumerical, poolsize, hipMemcpyHostToDevice );
  hipMemcpy( cudahashPassword, &hashPassword, hashPwdsize, hipMemcpyHostToDevice );
  printf("======Start cracking using Cuda======\n");
  double minCuda = 1e30;
  struct timespec start, stop;
  for(unsigned int i = 0 ; i < 3; i ++){
     //reset_and_start_timer();
    clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &start);
     for(int length=1;length<=maxLen;length++){
     //int length = 5;
      cudaPermutation<<<1,36>>>(length,0,cudahashPassword,length,output,cudaalphanumerical);
      hipDeviceSynchronize();
     }

     //double dt = get_elapsed_mcycles();
     clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &stop);
     double dt = (stop.tv_sec - start.tv_sec) * 1e6 + (stop.tv_nsec - start.tv_nsec) / 1e3;    // in microseconds
     minCuda = std::min(minCuda, dt);
   }
  printf("@time of minCuda run:\t\t\t[%.3f] microseconds\n", minCuda);
  return 0;
}